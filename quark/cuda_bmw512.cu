#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

//#define SHL(x, n)            ((x) << (n))
//#define SHR(x, n)            ((x) >> (n))
#define SHR(x, n) SHR2(x, n) 
#define SHL(x, n) SHL2(x, n) 

#undef ROTL64
#define ROTL64 ROL2


#define CONST_EXP2    q[i+0] + ROTL64(q[i+1], 5)  + q[i+2] + ROTL64(q[i+3], 11) + \
                    q[i+4] + ROTL64(q[i+5], 27) + q[i+6] + SWAPDWORDS2(q[i+7]) + \
                    q[i+8] + ROTL64(q[i+9], 37) + q[i+10] + ROTL64(q[i+11], 43) + \
                    q[i+12] + ROTL64(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

__device__ void Compression512_64_first(uint2 *msg, uint2 *hash)
{
	// Compression ref. implementation
	uint2 q[32];
	uint2 tmp;

	tmp = (msg[5] ^ hash[5]) - (msg[7] ^ hash[7]) + (hash[10]) + (hash[13]) + (hash[14]);
	q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[1];
	tmp = (msg[6] ^ hash[6]) - (msg[8] ^ hash[8]) + (hash[11]) + (hash[14]) - (msg[15] ^ hash[15]);
	q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
	tmp = (msg[0] ^ hash[0]) + (msg[7] ^ hash[7]) + (hash[9]) - (hash[12]) + (msg[15] ^ hash[15]);
	q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
	tmp = (msg[0] ^ hash[0]) - (msg[1] ^ hash[1]) + (msg[8] ^ hash[8]) - (hash[10]) + (hash[13]);
	q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
	tmp = (msg[1] ^ hash[1]) + (msg[2] ^ hash[2]) + (hash[9]) - (hash[11]) - (hash[14]);
	q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
	tmp = (msg[3] ^ hash[3]) - (msg[2] ^ hash[2]) + (hash[10]) - (hash[12]) + (msg[15] ^ hash[15]);
	q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[6];
	tmp = (msg[4] ^ hash[4]) - (msg[0] ^ hash[0]) - (msg[3] ^ hash[3]) - (hash[11]) + (hash[13]);
	q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
	tmp = (msg[1] ^ hash[1]) - (msg[4] ^ hash[4]) - (msg[5] ^ hash[5]) - (hash[12]) - (hash[14]);
	q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
	tmp = (msg[2] ^ hash[2]) - (msg[5] ^ hash[5]) - (msg[6] ^ hash[6]) + (hash[13]) - (msg[15] ^ hash[15]);
	q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
	tmp = (msg[0] ^ hash[0]) - (msg[3] ^ hash[3]) + (msg[6] ^ hash[6]) - (msg[7] ^ hash[7]) + (hash[14]);
	q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
	tmp = (msg[8] ^ hash[8]) - (msg[1] ^ hash[1]) - (msg[4] ^ hash[4]) - (msg[7] ^ hash[7]) + (msg[15] ^ hash[15]);
	q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[11];
	tmp = (msg[8] ^ hash[8]) - (msg[0] ^ hash[0]) - (msg[2] ^ hash[2]) - (msg[5] ^ hash[5]) + (hash[9]);
	q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
	tmp = (msg[1] ^ hash[1]) + (msg[3] ^ hash[3]) - (msg[6] ^ hash[6]) - (hash[9]) + (hash[10]);
	q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
	tmp = (msg[2] ^ hash[2]) + (msg[4] ^ hash[4]) + (msg[7] ^ hash[7]) + (hash[10]) + (hash[11]);
	q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
	tmp = (msg[3] ^ hash[3]) - (msg[5] ^ hash[5]) + (msg[8] ^ hash[8]) - (hash[11]) - (hash[12]);
	q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
	tmp = (msg[12] ^ hash[12]) - (msg[4] ^ hash[4]) - (msg[6] ^ hash[6]) - (hash[9]) + (hash[13]);
	q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

	// Expand 1
#pragma unroll 2
	for (int i = 0; i<2; i++)
	{
		q[i + 16] =
			(SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64(q[i], 13) ^ ROTL64(q[i], 43)) +
			(SHR(q[i + 1], 2) ^ SHL(q[i + 1], 1) ^ ROTL64(q[i + 1], 19) ^ ROTL64(q[i + 1], 53)) +
			(SHR(q[i + 2], 2) ^ SHL(q[i + 2], 2) ^ ROTL64(q[i + 2], 28) ^ ROTL64(q[i + 2], 59)) +
			(SHR(q[i + 3], 1) ^ SHL(q[i + 3], 3) ^ ROTL64(q[i + 3], 4) ^ ROTL64(q[i + 3], 37)) +
			(SHR(q[i + 4], 1) ^ SHL(q[i + 4], 2) ^ ROTL64(q[i + 4], 13) ^ ROTL64(q[i + 4], 43)) +
			(SHR(q[i + 5], 2) ^ SHL(q[i + 5], 1) ^ ROTL64(q[i + 5], 19) ^ ROTL64(q[i + 5], 53)) +
			(SHR(q[i + 6], 2) ^ SHL(q[i + 6], 2) ^ ROTL64(q[i + 6], 28) ^ ROTL64(q[i + 6], 59)) +
			(SHR(q[i + 7], 1) ^ SHL(q[i + 7], 3) ^ ROTL64(q[i + 7], 4) ^ ROTL64(q[i + 7], 37)) +
			(SHR(q[i + 8], 1) ^ SHL(q[i + 8], 2) ^ ROTL64(q[i + 8], 13) ^ ROTL64(q[i + 8], 43)) +
			(SHR(q[i + 9], 2) ^ SHL(q[i + 9], 1) ^ ROTL64(q[i + 9], 19) ^ ROTL64(q[i + 9], 53)) +
			(SHR(q[i + 10], 2) ^ SHL(q[i + 10], 2) ^ ROTL64(q[i + 10], 28) ^ ROTL64(q[i + 10], 59)) +
			(SHR(q[i + 11], 1) ^ SHL(q[i + 11], 3) ^ ROTL64(q[i + 11], 4) ^ ROTL64(q[i + 11], 37)) +
			(SHR(q[i + 12], 1) ^ SHL(q[i + 12], 2) ^ ROTL64(q[i + 12], 13) ^ ROTL64(q[i + 12], 43)) +
			(SHR(q[i + 13], 2) ^ SHL(q[i + 13], 1) ^ ROTL64(q[i + 13], 19) ^ ROTL64(q[i + 13], 53)) +
			(SHR(q[i + 14], 2) ^ SHL(q[i + 14], 2) ^ ROTL64(q[i + 14], 28) ^ ROTL64(q[i + 14], 59)) +
			(SHR(q[i + 15], 1) ^ SHL(q[i + 15], 3) ^ ROTL64(q[i + 15], 4) ^ ROTL64(q[i + 15], 37)) +
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
			ROTL64(msg[i + 3], i + 4)) ^ hash[i + 7]);
	}

#pragma unroll 4
	for (int i = 2; i<6; i++) {
		q[i + 16] = CONST_EXP2 +
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
			ROTL64(msg[i + 3], i + 4) - ROTL64(msg[i + 10], i + 11)) ^ hash[i + 7]);
	}
#pragma unroll 3
	for (int i = 6; i<9; i++) {
		q[i + 16] = CONST_EXP2 +
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) -
			ROTL64(msg[i - 6], (i - 6) + 1)) ^ hash[i + 7]);
	}
#pragma unroll 4
	for (int i = 9; i<13; i++) {
		q[i + 16] = CONST_EXP2 +
			((vectorize((i + 16)*(0x0555555555555555ull)) +
			ROTL64(msg[i + 3], i + 4) - ROTL64(msg[i - 6], (i - 6) + 1)) ^ hash[i - 9]);
	}
#pragma unroll 3
	for (int i = 13; i<16; i++) {
		q[i + 16] = CONST_EXP2 +
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
			ROTL64(msg[i - 13], (i - 13) + 1) - ROTL64(msg[i - 6], (i - 6) + 1)) ^ hash[i - 9]);
	}

	uint2 XL64 = q[16] ^ q[17] ^ q[18] ^ q[19] ^ q[20] ^ q[21] ^ q[22] ^ q[23];
	uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

	hash[0] = (SHL(XH64, 5) ^ SHR(q[16], 5) ^ msg[0]) + (XL64    ^ q[24] ^ q[0]);
	hash[1] = (SHR(XH64, 7) ^ SHL(q[17], 8) ^ msg[1]) + (XL64    ^ q[25] ^ q[1]);
	hash[2] = (SHR(XH64, 5) ^ SHL(q[18], 5) ^ msg[2]) + (XL64    ^ q[26] ^ q[2]);
	hash[3] = (SHR(XH64, 1) ^ SHL(q[19], 5) ^ msg[3]) + (XL64    ^ q[27] ^ q[3]);
	hash[4] = (SHR(XH64, 3) ^ q[20] ^ msg[4]) + (XL64    ^ q[28] ^ q[4]);
	hash[5] = (SHL(XH64, 6) ^ SHR(q[21], 6) ^ msg[5]) + (XL64    ^ q[29] ^ q[5]);
	hash[6] = (SHR(XH64, 4) ^ SHL(q[22], 6) ^ msg[6]) + (XL64    ^ q[30] ^ q[6]);
	hash[7] = (SHR(XH64, 11) ^ SHL(q[23], 2) ^ msg[7]) + (XL64    ^ q[31] ^ q[7]);

	hash[8] = ROTL64(hash[4], 9) + (XH64     ^     q[24] ^ msg[8]) + (SHL(XL64, 8) ^ q[23] ^ q[8]);
	hash[9] = ROTL64(hash[5], 10) + (XH64     ^     q[25]) + (SHR(XL64, 6) ^ q[16] ^ q[9]);
	hash[10] = ROTL64(hash[6], 11) + (XH64     ^     q[26]) + (SHL(XL64, 6) ^ q[17] ^ q[10]);
	hash[11] = ROTL64(hash[7], 12) + (XH64     ^     q[27]) + (SHL(XL64, 4) ^ q[18] ^ q[11]);
	hash[12] = ROTL64(hash[0], 13) + (XH64     ^     q[28]) + (SHR(XL64, 3) ^ q[19] ^ q[12]);
	hash[13] = ROTL64(hash[1], 14) + (XH64     ^     q[29]) + (SHR(XL64, 4) ^ q[20] ^ q[13]);
	hash[14] = ROTL64(hash[2], 15) + (XH64     ^     q[30]) + (SHR(XL64, 7) ^ q[21] ^ q[14]);
	hash[15] = ROTL64(hash[3], 16) + (XH64     ^     q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);
}

__device__ void Compression512(uint2 *msg, uint2 *hash)
{
    // Compression ref. implementation
	uint2 q[32];
	uint2 tmp;

    tmp = (msg[ 5] ^ hash[ 5]) - (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]) + (msg[14] ^ hash[14]);
    q[0] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[1];
    tmp = (msg[ 6] ^ hash[ 6]) - (msg[ 8] ^ hash[ 8]) + (msg[11] ^ hash[11]) + (msg[14] ^ hash[14]) - (msg[15] ^ hash[15]);
    q[1] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[2];
    tmp = (msg[ 0] ^ hash[ 0]) + (msg[ 7] ^ hash[ 7]) + (msg[ 9] ^ hash[ 9]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[2] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[3];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 1] ^ hash[ 1]) + (msg[ 8] ^ hash[ 8]) - (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]);
    q[3] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[4];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 2] ^ hash[ 2]) + (msg[ 9] ^ hash[ 9]) - (msg[11] ^ hash[11]) - (msg[14] ^ hash[14]);
    q[4] = (SHR(tmp, 1) ^ tmp) + hash[5];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 2] ^ hash[ 2]) + (msg[10] ^ hash[10]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    q[5] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[6];
    tmp = (msg[ 4] ^ hash[ 4]) - (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) - (msg[11] ^ hash[11]) + (msg[13] ^ hash[13]);
    q[6] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[7];
    tmp = (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 5] ^ hash[ 5]) - (msg[12] ^ hash[12]) - (msg[14] ^ hash[14]);
    q[7] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[8];
    tmp = (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) - (msg[ 6] ^ hash[ 6]) + (msg[13] ^ hash[13]) - (msg[15] ^ hash[15]);
    q[8] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[9];
    tmp = (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) + (msg[ 6] ^ hash[ 6]) - (msg[ 7] ^ hash[ 7]) + (msg[14] ^ hash[14]);
    q[9] = (SHR(tmp, 1) ^ tmp) + hash[10];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 7] ^ hash[ 7]) + (msg[15] ^ hash[15]);
    q[10] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp,  4) ^ ROTL64(tmp, 37)) + hash[11];
    tmp = (msg[ 8] ^ hash[ 8]) - (msg[ 0] ^ hash[ 0]) - (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) + (msg[ 9] ^ hash[ 9]);
    q[11] = (SHR(tmp, 1) ^ SHL(tmp, 2) ^ ROTL64(tmp, 13) ^ ROTL64(tmp, 43)) + hash[12];
    tmp = (msg[ 1] ^ hash[ 1]) + (msg[ 3] ^ hash[ 3]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[10] ^ hash[10]);
    q[12] = (SHR(tmp, 2) ^ SHL(tmp, 1) ^ ROTL64(tmp, 19) ^ ROTL64(tmp, 53)) + hash[13];
    tmp = (msg[ 2] ^ hash[ 2]) + (msg[ 4] ^ hash[ 4]) + (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[11] ^ hash[11]);
    q[13] = (SHR(tmp, 2) ^ SHL(tmp, 2) ^ ROTL64(tmp, 28) ^ ROTL64(tmp, 59)) + hash[14];
    tmp = (msg[ 3] ^ hash[ 3]) - (msg[ 5] ^ hash[ 5]) + (msg[ 8] ^ hash[ 8]) - (msg[11] ^ hash[11]) - (msg[12] ^ hash[12]);
    q[14] = (SHR(tmp, 1) ^ tmp) + hash[15];
    tmp = (msg[12] ^ hash[12]) - (msg[ 4] ^ hash[ 4]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[13] ^ hash[13]);
    q[15] = (SHR(tmp, 1) ^ SHL(tmp, 3) ^ ROTL64(tmp, 4) ^ ROTL64(tmp, 37)) + hash[0];

    // Expand 1
#pragma unroll 2
    for(int i=0;i<2;i++)
    {
        q[i+16] =
        (SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64(q[i], 13) ^ ROTL64(q[i], 43)) +
        (SHR(q[i+1], 2) ^ SHL(q[i+1], 1) ^ ROTL64(q[i+1], 19) ^ ROTL64(q[i+1], 53)) +
        (SHR(q[i+2], 2) ^ SHL(q[i+2], 2) ^ ROTL64(q[i+2], 28) ^ ROTL64(q[i+2], 59)) +
        (SHR(q[i+3], 1) ^ SHL(q[i+3], 3) ^ ROTL64(q[i+3],  4) ^ ROTL64(q[i+3], 37)) +
        (SHR(q[i+4], 1) ^ SHL(q[i+4], 2) ^ ROTL64(q[i+4], 13) ^ ROTL64(q[i+4], 43)) +
        (SHR(q[i+5], 2) ^ SHL(q[i+5], 1) ^ ROTL64(q[i+5], 19) ^ ROTL64(q[i+5], 53)) +
        (SHR(q[i+6], 2) ^ SHL(q[i+6], 2) ^ ROTL64(q[i+6], 28) ^ ROTL64(q[i+6], 59)) +
        (SHR(q[i+7], 1) ^ SHL(q[i+7], 3) ^ ROTL64(q[i+7],  4) ^ ROTL64(q[i+7], 37)) +
        (SHR(q[i+8], 1) ^ SHL(q[i+8], 2) ^ ROTL64(q[i+8], 13) ^ ROTL64(q[i+8], 43)) +
        (SHR(q[i+9], 2) ^ SHL(q[i+9], 1) ^ ROTL64(q[i+9], 19) ^ ROTL64(q[i+9], 53)) +
        (SHR(q[i+10], 2) ^ SHL(q[i+10], 2) ^ ROTL64(q[i+10], 28) ^ ROTL64(q[i+10], 59)) +
        (SHR(q[i+11], 1) ^ SHL(q[i+11], 3) ^ ROTL64(q[i+11],  4) ^ ROTL64(q[i+11], 37)) +
        (SHR(q[i+12], 1) ^ SHL(q[i+12], 2) ^ ROTL64(q[i+12], 13) ^ ROTL64(q[i+12], 43)) +
        (SHR(q[i+13], 2) ^ SHL(q[i+13], 1) ^ ROTL64(q[i+13], 19) ^ ROTL64(q[i+13], 53)) +
        (SHR(q[i+14], 2) ^ SHL(q[i+14], 2) ^ ROTL64(q[i+14], 28) ^ ROTL64(q[i+14], 59)) +
        (SHR(q[i+15], 1) ^ SHL(q[i+15], 3) ^ ROTL64(q[i+15],  4) ^ ROTL64(q[i+15], 37)) +
        ((    vectorize((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i+10], i+11) ) ^ hash[i+7]);
    }

#pragma unroll 4
    for(int i=2;i<6;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    vectorize((i+16)*(0x0555555555555555ull)) + ROTL64(msg[i], i+1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i+10], i+11) ) ^ hash[i+7]);
    }
#pragma unroll 3
    for(int i=6;i<9;i++) {
        q[i+16] = CONST_EXP2 + 
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i+7]);
    }
#pragma unroll 4
    for(int i=9;i<13;i++) {
        q[i+16] = CONST_EXP2 + 
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
            ROTL64(msg[i+3], i+4) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i-9]);
    }
#pragma unroll 3
    for(int i=13;i<16;i++) {
        q[i+16] = CONST_EXP2 + 
			((vectorize((i + 16)*(0x0555555555555555ull)) + ROTL64(msg[i], i + 1) +
            ROTL64(msg[i-13], (i-13)+1) - ROTL64(msg[i-6], (i-6)+1) ) ^ hash[i-9]);
    }

    uint2 XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
	uint2 XH64 = XL64^q[24] ^ q[25] ^ q[26] ^ q[27] ^ q[28] ^ q[29] ^ q[30] ^ q[31];

    hash[0] =                       (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[ 0]) + (    XL64    ^ q[24] ^ q[ 0]);
    hash[1] =                       (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[ 1]) + (    XL64    ^ q[25] ^ q[ 1]);
    hash[2] =                       (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[ 2]) + (    XL64    ^ q[26] ^ q[ 2]);
    hash[3] =                       (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[ 3]) + (    XL64    ^ q[27] ^ q[ 3]);
    hash[4] =                       (SHR(XH64, 3) ^     q[20]    ^ msg[ 4]) + (    XL64    ^ q[28] ^ q[ 4]);
    hash[5] =                       (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[ 5]) + (    XL64    ^ q[29] ^ q[ 5]);
    hash[6] =                       (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[ 6]) + (    XL64    ^ q[30] ^ q[ 6]);
    hash[7] =                       (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[ 7]) + (    XL64    ^ q[31] ^ q[ 7]);

    hash[ 8] = ROTL64(hash[4], 9) + (    XH64     ^     q[24]    ^ msg[ 8]) + (SHL(XL64,8) ^ q[23] ^ q[ 8]);
    hash[ 9] = ROTL64(hash[5],10) + (    XH64     ^     q[25]    ^ msg[ 9]) + (SHR(XL64,6) ^ q[16] ^ q[ 9]);
    hash[10] = ROTL64(hash[6],11) + (    XH64     ^     q[26]    ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
    hash[11] = ROTL64(hash[7],12) + (    XH64     ^     q[27]    ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
    hash[12] = ROTL64(hash[0],13) + (    XH64     ^     q[28]    ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
    hash[13] = ROTL64(hash[1],14) + (    XH64     ^     q[29]    ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
    hash[14] = ROTL64(hash[2],15) + (    XH64     ^     q[30]    ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
	hash[15] = ROTL64(hash[3],16) + (XH64     ^     q[31] ^ msg[15]) + (SHR(XL64, 2) ^ q[22] ^ q[15]);
}

__global__ __launch_bounds__(256, 2)
void quark_bmw512_gpu_hash_64(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint64_t *inpHash = &g_hash[8 * hashPosition];

        // Init
		uint2 h[16] = {
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL, },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL, },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};
        // Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
        // BMW arbeitet mit 128 Byte!!!
		uint2 message[16];
#pragma unroll 8
        for(int i=0;i<8;i++)
			message[i] = vectorize(inpHash[i]);
#pragma unroll 6
        for(int i=9;i<15;i++)
            message[i] = make_uint2(0,0);

        // Padding einf�gen (Byteorder?!?)
		message[8] = make_uint2(0x80,0);
        // L�nge (in Bits, d.h. 64 Byte * 8 = 512 Bits
		message[15] = make_uint2(512,0);

        // Compression 1
        Compression512_64_first(message, h);

        // Final
#pragma unroll 16
        for(int i=0;i<16;i++)
		{
			message[i].y = 0xaaaaaaaa;
			message[i].x = 0xaaaaaaa0ul + (uint32_t)i;
		}
        Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = devectorize(message[i+8]);
    }
}

__global__ __launch_bounds__(256, 2)
void quark_bmw512_gpu_hash_80(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
    uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = startNounce + thread;

        // Init
		uint2 h[16] = {
			{ 0x84858687UL, 0x80818283UL },
			{ 0x8C8D8E8FUL, 0x88898A8BUL },
			{ 0x94959697UL, 0x90919293UL },
			{ 0x9C9D9E9FUL, 0x98999A9BUL },
			{ 0xA4A5A6A7UL, 0xA0A1A2A3UL },
			{ 0xACADAEAFUL, 0xA8A9AAABUL },
			{ 0xB4B5B6B7UL, 0xB0B1B2B3UL },
			{ 0xBCBDBEBFUL, 0xB8B9BABBUL },
			{ 0xC4C5C6C7UL, 0xC0C1C2C3UL, },
			{ 0xCCCDCECFUL, 0xC8C9CACBUL, },
			{ 0xD4D5D6D7UL, 0xD0D1D2D3UL },
			{ 0xDCDDDEDFUL, 0xD8D9DADBUL },
			{ 0xE4E5E6E7UL, 0xE0E1E2E3UL },
			{ 0xECEDEEEFUL, 0xE8E9EAEBUL },
			{ 0xF4F5F6F7UL, 0xF0F1F2F3UL },
			{ 0xFCFDFEFFUL, 0xF8F9FAFBUL }
		};
        // Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
        // BMW arbeitet mit 128 Byte!!!
		uint2 message[16];
#pragma unroll 16
        for(int i=0;i<16;i++)
			message[i] = vectorize(c_PaddedMessage80[i]);

        // die Nounce durch die thread-spezifische ersetzen
		message[9].x = cuda_swab32(nounce);	//REPLACE_HIWORD(message[9], cuda_swab32(nounce));
        // Compression 1
        Compression512(message, h);

        // Final
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = vectorize(0xaaaaaaaaaaaaaaa0ull + (uint64_t)i);

        Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * thread];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = devectorize(message[i+8]);
    }
}

// Setup-Funktionen
__host__ void quark_bmw512_cpu_init(int thr_id, uint32_t threads)
{
}

// Bmw512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_bmw512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	uint64_t *message = (uint64_t*)PaddedMessage;
	// Padding einf�gen (Byteorder?!?)
	message[10] = SPH_C64(0x80);
	// L�nge (in Bits, d.h. 80 Byte * 8 = 640 Bits
	message[15] = SPH_C64(640);

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol(HIP_SYMBOL( c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void quark_bmw512_cpu_hash_64(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 32;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_bmw512_gpu_hash_64<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    MyStreamSynchronize(NULL, order, thr_id);
}

__host__ void quark_bmw512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order)
{
    const uint32_t threadsperblock = 128;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    quark_bmw512_gpu_hash_80<<<grid, block>>>(threads, startNounce, (uint64_t*)d_hash);
    MyStreamSynchronize(NULL, order, thr_id);
}

