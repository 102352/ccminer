#include "hip/hip_runtime.h"
extern "C"
{
#include "sph/neoscrypt.h"
}

#include "cuda_helper.h"
#include "miner.h"
#include <string.h>
static uint32_t *d_hash[MAX_GPUS], hw_errors = 0;
static uint32_t *foundNonce;

extern void neoscrypt_setBlockTarget(int thr_id, uint32_t * data, const void *ptarget);
extern void neoscrypt_cpu_init(int thr_id, uint32_t threads,uint32_t* hash);
extern void neoscrypt_cpu_hash_k4(int stratum, int thr_id, int threads, uint32_t startNounce, int threadsperblock, uint32_t* foundnonce);
//extern void neoscrypt_cpu_hash_k4_52(int stratum, int thr_id, int threads, uint32_t startNounce, int order, uint32_t* foundnonce);


int scanhash_neoscrypt(bool stratum, int thr_id, uint32_t *pdata,
    uint32_t *ptarget, uint32_t max_nonce,
    uint32_t *hashes_done)
	const uint32_t first_nonce = pdata[19];
	int intensity = (256 * 64 * 3);
	static uint32_t throughput;
	static volatile bool init[MAX_GPUS] = { false };


	if (opt_benchmark) {
		((uint32_t*)ptarget)[7] = 0x01ff;
		stratum = 0;
	}
	
	if (!init[thr_id]) {
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device_map[thr_id]);
	unsigned int cc = props.major * 10 + props.minor;
	if(cc < 32)
	{
		applog(LOG_ERR, "GPU #%d: this gpu is not supported", device_map[thr_id]);
		mining_has_stopped[thr_id] = true;
		proper_exit(2);
	}
		
		if      (strstr(props.name, "970"))    intensity = (256 * 64 * 4);
		else if (strstr(props.name, "980"))    intensity = (256 * 64 * 4);
		else if (strstr(props.name, "750 Ti")) intensity = (256 * 32 * 7);
		else if (strstr(props.name, "750"))    intensity = (256 * 32 * 7 / 2);
		else if (strstr(props.name, "960"))    intensity = (256 * 32 * 7);



	uint32_t throughput = device_intensity(device_map[thr_id], __func__, intensity);

	throughput = min(throughput, (max_nonce - first_nonce));

		CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
//		hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
//		hipDeviceSetCacheConfig(hipFuncCachePreferL1);	
		CUDA_SAFE_CALL(hipStreamCreate(&gpustream[thr_id]));
		CUDA_SAFE_CALL(hipHostMalloc(&foundNonce, 2 * 4));
		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 32 * 130 * sizeof(uint64_t) * throughput));
		neoscrypt_cpu_init(thr_id, d_hash[thr_id]);
		init[thr_id] = true;
	}

	uint32_t endiandata[20];
	for (int k = 0; k < 20; k++) { 
		if (stratum) be32enc(&endiandata[k], ((uint32_t*)pdata)[k]);
		else endiandata[k] = pdata[k];
	}
	neoscrypt_setBlockTarget(thr_id, endiandata, ptarget);

	do {
//		int order = 0;
		uint32_t foundNonce;
		neoscrypt_cpu_hash_k4(stratum, thr_id, throughput, pdata[19], foundNonce, (device_sm[device_map[thr_id]] > 500 ? 128 : 32));
		if(stop_mining) {mining_has_stopped[thr_id] = true; hipStreamDestroy(gpustream[thr_id]); pthread_exit(nullptr);}
		if(foundNonce[0] != 0xffffffff)

		if  (foundNonce != 0xffffffff && foundNonce != 0x0) {
			if (opt_benchmark) applog(LOG_INFO, "GPU #%d Found nounce %08x", thr_id, foundNonce);
			uint32_t vhash64[8];

			if (stratum) be32enc(&endiandata[19], foundNonce[0]);
			else endiandata[19] = foundNonce;
			neoscrypt((unsigned char*)endiandata, (unsigned char*)vhash64, 0x80000620);
			*hashes_done = foundNonce - first_nonce + 1;

			if (hw_errors > 0) applog(LOG_INFO, "Hardware errors: %u", hw_errors);

			if(vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget))
			{
				int res = 1;
				*hashes_done = pdata[19] - first_nonce + throughput;
				if(foundNonce[1] != 0xffffffff)
				{
					if(stratum)
					{
						be32enc(&endiandata[19], foundNonce[1]);
					}
					else
					{
						endiandata[19] = foundNonce[1];
					}
					neoscrypt((unsigned char*)endiandata, (unsigned char*)vhash64, 0x80000620);
					if(vhash64[7] <= ptarget[7] && fulltest(vhash64, ptarget))
					{

						pdata[21] = foundNonce[1];
						res++;
						if(opt_benchmark)
							applog(LOG_INFO, "GPU #%d: Found second nounce %08x", device_map[thr_id], foundNonce[1]);
					}
					else
					{
						if(vhash64[7] != ptarget[7])
						{
							applog(LOG_WARNING, "GPU #%d: result for %08x does not validate on CPU!", device_map[thr_id], foundNonce[1]);
						}
					}

				}
				pdata[19] = foundNonce[0];
				if(opt_benchmark)
					applog(LOG_INFO, "GPU #%d: Found nounce %08x", device_map[thr_id], foundNonce[0]);
				return res;
			}
			else
			{
				if(vhash64[7] != ptarget[7])
				   applog(LOG_WARNING, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}
		pdata[19] += throughput;
} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}

