#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>

#include "cuda_helper.h"

// aus heavy.cu
extern hipError_t MyStreamSynchronize(hipStream_t stream, int situation, int thr_id);

#include "cuda_x11_aes.cu"

__device__ __forceinline__ void AES_2ROUND(
	const uint32_t* __restrict__ sharedMemory,
	uint32_t &x0, uint32_t &x1, uint32_t &x2, uint32_t &x3,
	uint32_t &k0)
{
	uint32_t y0, y1, y2, y3;
	
	aes_round(sharedMemory,
		x0, x1, x2, x3,
		k0,
		y0, y1, y2, y3);

	aes_round(sharedMemory,
		y0, y1, y2, y3,
		x0, x1, x2, x3);

	// hier werden wir ein carry brauchen (oder auch nicht)
	k0++;
}


__device__ __forceinline__ void cuda_echo_round(
	const uint32_t *const __restrict__ sharedMemory, uint32_t *const __restrict__  hash)
{
	uint32_t k0;
	uint32_t h[16];

	const uint32_t P[48] = {
		0xe7e9f5f5,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		0xa4213d7e,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,
		//8-12
		0x01425eb8,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		0x65978b09,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		//21-25
		0x2cb6b661,
		0x6b23b3b3,
		0xcf93a7cf,
		0x9d9d3751,

		0x9ac2dea3,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		//34-38
		0x579f9f33,
		0xfbfbfbfb,
		0xfbfbfbfb,
		0xefefd3c7,

		0xdbfde1dd,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		0x34514d9e,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,


		0xb134347e,
		0xea6f7e7e,
		0xbd7731bd,
		0x8a8a1968,

		0x14b8a457,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af,

		0x265f4382,
		0xf5e7e9f5,
		0xb3b36b23,
		0xb3dbe7af
		//58-61
	};

#pragma unroll
	for (int i = 0; i < 16; i++)
	{
		h[i] = hash[i];
	}

	k0 = 512 + 8;

#pragma unroll
	for (int idx = 0; idx < 16; idx+= 4)
	{
		AES_2ROUND(sharedMemory,
			h[idx + 0], h[idx + 1], h[idx + 2], h[idx + 3], k0);
	}
	k0 += 4;

	uint32_t W[64];

#pragma unroll
	for (int i = 0; i < 4; i++) 
	{
		
		const uint32_t ab = P[i] ^ P[i + 4];
		const uint32_t bc = P[i + 4] ^ h[i + 8];
		const uint32_t cd = h[i + 8] ^ P[i + 8];

		const uint32_t t = (ab & 0x80808080);
		const uint32_t t2 = (bc & 0x80808080);
		const uint32_t t3 = (cd & 0x80808080);

		const uint32_t abx = (t >> 7) * 27 ^ ((ab^t) << 1);
		const uint32_t bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
		const uint32_t cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

		W[0 + i] = abx ^ bc ^ P[i + 8];
		W[0 + i + 4] = bcx ^ P[i] ^ cd;
		W[0 + i + 8] = cdx ^ ab ^ P[i + 8];
		W[0 + i + 12] = abx ^ bcx ^ cdx ^ ab ^ h[i + 8];

		const uint32_t ab_2 = P[12 + i] ^ h[i + 4];
		const uint32_t bc_2 = h[i + 4] ^ P[12 + i + 4];
		const uint32_t cd_2 = P[12 + i + 4] ^ P[12 + i + 8];


		const uint32_t t_2 = (ab_2 & 0x80808080);
		const uint32_t t2_2 = (bc_2 & 0x80808080);
		const uint32_t t3_2 = (cd_2 & 0x80808080);

		const uint32_t abx_2 = (t_2 >> 7) * 27 ^ ((ab_2^t_2) << 1);
		const uint32_t bcx_2 = (t2_2 >> 7) * 27 ^ ((bc_2^t2_2) << 1);
		const uint32_t cdx_2 = (t3_2 >> 7) * 27 ^ ((cd_2^t3_2) << 1);

		W[16 + i] = abx_2 ^ bc_2 ^ P[12 + i + 8];
		W[16 + i + 4] = bcx_2 ^ P[12 + i] ^ cd_2;
		W[16 + i + 8] = cdx_2 ^ ab_2 ^ P[12 + i + 8];
		W[16 + i + 12] = abx_2 ^ bcx_2 ^ cdx_2 ^ ab_2 ^ P[12 + i + 4];


		const uint32_t ab_3 = h[i] ^ P[24 + i + 0];
		const uint32_t bc_3 = P[24 + i + 0] ^ P[24 + i + 4];
		const uint32_t cd_3 = P[24 + i + 4] ^ P[24 + i + 8];

		const uint32_t t_3 = (ab_3 & 0x80808080);
		const uint32_t t2_3 = (bc_3 & 0x80808080);
		const uint32_t t3_3 = (cd_3 & 0x80808080);

		const uint32_t abx_3 = (t_3 >> 7) * 27 ^ ((ab_3^t_3) << 1);
		const uint32_t bcx_3 = (t2_3 >> 7) * 27 ^ ((bc_3^t2_3) << 1);
		const uint32_t cdx_3 = (t3_3 >> 7) * 27 ^ ((cd_3^t3_3) << 1);

		W[32 + i] = abx_3 ^ bc_3 ^ P[24 + i + 8];
		W[32 + i + 4] = bcx_3 ^ h[i] ^ cd_3;
		W[32 + i + 8] = cdx_3 ^ ab_3 ^ P[24 + i + 8];
		W[32 + i + 12] = abx_3 ^ bcx_3 ^ cdx_3 ^ ab_3 ^ P[24 + i + 4];

		const uint32_t ab_4 = P[36 + i] ^ P[36 + i + 4];
		const uint32_t bc_4 = P[36 + i + 4] ^ P[36 + i + 8];
		const uint32_t cd_4 = P[36 + i + 8] ^ h[i + 12];

		const uint32_t t_4 = (ab_4 & 0x80808080);
		const uint32_t t2_4 = (bc_4 & 0x80808080);
		const uint32_t t3_4 = (cd_4 & 0x80808080);

		const uint32_t abx_4 = (t_4 >> 7) * 27 ^ ((ab_4^t_4) << 1);
		const uint32_t bcx_4 = (t2_4 >> 7) * 27 ^ ((bc_4^t2_4) << 1);
		const uint32_t cdx_4 = (t3_4 >> 7) * 27 ^ ((cd_4^t3_4) << 1);

		W[48 + i] = abx_4 ^ bc_4 ^ h[i + 12];
		W[48 + i + 4] = bcx_4 ^ P[36 + i] ^ cd_4;
		W[48 + i + 8] = cdx_4 ^ ab_4 ^ h[i + 12];
		W[48 + i + 12] = abx_4 ^ bcx_4 ^ cdx_4 ^ ab_4 ^ P[36 + i + 8];
	}

	for (int k = 1; k < 10; k++)
	{

		// Big Sub Words
		#pragma unroll
		for (int idx = 0; idx < 64; idx+=16)
		{
			AES_2ROUND(sharedMemory,
				W[idx + 0], W[idx + 1], W[idx + 2], W[idx + 3],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 4], W[idx + 5], W[idx + 6], W[idx + 7],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 8], W[idx + 9], W[idx + 10], W[idx + 11],
				k0);
			AES_2ROUND(sharedMemory,
				W[idx + 12], W[idx + 13], W[idx + 14], W[idx + 15],
				k0);

		}

		// Shift Rows
#pragma unroll 4
		for (int i = 0; i < 4; i++)
		{
			uint32_t t;

			/// 1, 5, 9, 13
			t = W[4 + i];
			W[4 + i] = W[20 + i];
			W[20 + i] = W[36 + i];
			W[36 + i] = W[52 + i];
			W[52 + i] = t;

			// 2, 6, 10, 14
			t = W[8 + i];
			W[8 + i] = W[40 + i];
			W[40 + i] = t;
			t = W[24 + i];
			W[24 + i] = W[56 + i];
			W[56 + i] = t;

			// 15, 11, 7, 3
			t = W[60 + i];
			W[60 + i] = W[44 + i];
			W[44 + i] = W[28 + i];
			W[28 + i] = W[12 + i];
			W[12 + i] = t;
		}

		// Mix Columns
#pragma unroll
		for (int i = 0; i < 4; i++) // Schleife über je 2*uint32_t
		{
#pragma unroll
			for (int idx = 0; idx < 64; idx += 16) // Schleife über die elemnte
			{

				const uint32_t ab = W[idx + i] ^ W[idx + i + 4];
				const uint32_t bc = W[idx + i + 4] ^ W[idx + i + 8];
				const uint32_t cd = W[idx + i + 8] ^ W[idx + i + 12];

				const uint32_t t = (ab & 0x80808080);
				const uint32_t t2 = (bc & 0x80808080);
				const uint32_t t3 = (cd & 0x80808080);

				const uint32_t abx = (t >> 7) * 27 ^ ((ab^t) << 1);
				const uint32_t bcx = (t2 >> 7) * 27 ^ ((bc^t2) << 1);
				const uint32_t cdx = (t3 >> 7) * 27 ^ ((cd^t3) << 1);

				W[idx + i] = abx ^ bc ^ W[idx + i + 12];
				W[idx + i + 4] = bcx ^ W[idx + i] ^ cd;
				W[idx + i + 8] = cdx ^ ab ^ W[idx + i + 12];
				W[idx + i + 12] = abx ^ bcx ^ cdx ^ ab ^ W[idx + i + 8];
			}
		}
	}

#pragma unroll
	for (int i = 0; i<16; i += 4)
	{
		W[i] ^= W[32 + i] ^ 512;
		W[i + 1] ^= W[32 + i + 1];
		W[i + 2] ^= W[32 + i + 2];
		W[i + 3] ^= W[32 + i + 3];
	}

#pragma unroll
	for (int i = 0; i<16; i++)
		hash[i] ^= W[i];
}


__device__ __forceinline__
void echo_gpu_init(uint32_t *const __restrict__ sharedMemory)
{
	/* each thread startup will fill a uint32 */
	if (threadIdx.x < 128) {
		sharedMemory[threadIdx.x] = d_AES0[threadIdx.x];
		sharedMemory[threadIdx.x + 256] = d_AES1[threadIdx.x];
		sharedMemory[threadIdx.x + 512] = d_AES2[threadIdx.x];
		sharedMemory[threadIdx.x + 768] = d_AES3[threadIdx.x];

		sharedMemory[threadIdx.x + 64 * 2] = d_AES0[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 256] = d_AES1[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 512] = d_AES2[threadIdx.x + 64 * 2];
		sharedMemory[threadIdx.x + 64 * 2 + 768] = d_AES3[threadIdx.x + 64 * 2];
	}
}


__global__ __launch_bounds__(128, 7)
void x11_echo512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
	__shared__ uint32_t sharedMemory[1024];

	echo_gpu_init(sharedMemory);

	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint32_t *Hash = (uint32_t*)&g_hash[hashPosition<<3];

		cuda_echo_round(sharedMemory, Hash);


    }
}

// Setup-Funktionen
__host__ void x11_echo512_cpu_init(int thr_id, int threads)
{
	aes_cpu_init();
}

__host__ void x11_echo512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int threadsperblock = 128;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Größe des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

//    fprintf(stderr, "threads=%d, %d blocks, %d threads per block, %d bytes shared\n", threads, grid.x, block.x, shared_size);

    x11_echo512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);
    MyStreamSynchronize(NULL, order, thr_id);
}
