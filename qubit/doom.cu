#include "hip/hip_runtime.h"
/*
 * qubit algorithm
 *
 */
extern "C" {
#include "sph/sph_luffa.h"
}

#include "miner.h"

#include "cuda_helper.h"

static uint32_t *d_hash[MAX_GPUS];

extern void qubit_luffa512_cpu_init(int thr_id, uint32_t threads);
extern void qubit_luffa512_cpu_setBlock_80(void *pdata);
extern void qubit_luffa512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);
extern void qubit_luffa512_cpufinal_setBlock_80(void *pdata, const void *ptarget);
extern uint32_t qubit_luffa512_cpu_finalhash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash);

extern void doomhash(void *state, const void *input)
{
	// luffa512
	sph_luffa512_context ctx_luffa;

	uint8_t hash[64];

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, input, 80);
	sph_luffa512_close(&ctx_luffa, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { false };

extern int scanhash_doom(int thr_id, uint32_t *pdata,
	uint32_t *ptarget, uint32_t max_nonce,
	uint32_t *hashes_done)
{
	const uint32_t first_nonce = pdata[19];
	uint32_t endiandata[20];
	uint32_t throughput = device_intensity(device_map[thr_id], __func__, 1U << 22); // 256*256*8*8
	throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		ptarget[7] = 0x0000f;

	if (!init[thr_id])
	{
		if (thr_id%opt_n_gputhreads == 0)
		{
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
			hipDeviceReset();
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		}
		else
		{
			while (!init[thr_id - thr_id%opt_n_gputhreads])
			{
			}
			CUDA_SAFE_CALL(hipSetDevice(device_map[thr_id]));
		}

		CUDA_SAFE_CALL(hipMalloc(&d_hash[thr_id], 16 * sizeof(uint32_t) * throughput));

		qubit_luffa512_cpu_init(thr_id, (int) throughput);

		init[thr_id] = true;
	}

	for (int k=0; k < 20; k++)
		be32enc(&endiandata[k], pdata[k]);

	qubit_luffa512_cpufinal_setBlock_80((void*)endiandata,ptarget);

	do {

		uint32_t foundNonce = qubit_luffa512_cpu_finalhash_80(thr_id, (int) throughput, pdata[19], d_hash[thr_id]);
		if (foundNonce != UINT32_MAX)
		{
			const uint32_t Htarg = ptarget[7];
			uint32_t vhash64[8];
			be32enc(&endiandata[19], foundNonce);
			doomhash(vhash64, endiandata);

			if (vhash64[7] <= Htarg && fulltest(vhash64, ptarget)) {
				*hashes_done = min(max_nonce - first_nonce, (uint64_t) pdata[19] - first_nonce + throughput);
				pdata[19] = foundNonce;
				return 1;
			}
			else {
				applog(LOG_INFO, "GPU #%d: result for nonce $%08X does not validate on CPU!", device_map[thr_id], foundNonce);
			}
		}

		pdata[19] += throughput; CUDA_SAFE_CALL(hipGetLastError());
	} while (!work_restart[thr_id].restart && ((uint64_t)max_nonce > ((uint64_t)(pdata[19]) + (uint64_t)throughput)));

	*hashes_done = pdata[19] - first_nonce + 1;
	return 0;
}
